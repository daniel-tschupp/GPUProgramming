#include "kernel_utils.h"

void pinHostMemory(float* hostPtr, size_t byteSize){
    CudaSafeAPICall(hipHostRegister(hostPtr,static_cast<uint>(byteSize), hipHostRegisterPortable|hipHostRegisterMapped));
}
float* mallocDeviceMemory(size_t byteSize){
    float *devPtr = nullptr;
    CudaSafeAPICall(hipMalloc((void**)&devPtr, byteSize));
    return devPtr;
}

float* pinHostAndMallocDeviceMemory(float* ptr, size_t byteSize){
    float *devPtr = nullptr;
    CudaSafeAPICall(hipHostRegister(ptr,static_cast<uint>(byteSize), hipHostRegisterPortable|hipHostRegisterMapped));
    CudaSafeAPICall(hipMalloc((void**)&devPtr, byteSize));
    return devPtr;
}
void cpyAsyncToDevice(float* devPtr, float* hostPtr, size_t byteSize, hipStream_t* stream){
    CudaSafeAPICall(hipMemcpyAsync(devPtr, hostPtr, byteSize, H2D, *stream));
}

void cpyAsyncToHost(float* devPtr, float* hostPtr, size_t byteSize, hipStream_t* stream){
    CudaSafeAPICall(hipMemcpyAsync(hostPtr, devPtr, byteSize, D2H, *stream));
}

void unpinHostMemory(float* hostPtr){
    CudaSafeAPICall(hipHostUnregister(hostPtr));
}
void freeDeviceMemory(float* devPtr){
    CudaSafeAPICall(hipFree(devPtr));
}
void unpinHostAndFreeDeviceMemory(float* devPtr, float* hostPtr){
    CudaSafeAPICall(hipHostUnregister(hostPtr));
    CudaSafeAPICall(hipFree(devPtr));
}

void waitForStream(hipStream_t* stream){
    CudaSafeAPICall(hipStreamSynchronize(*stream));
}

hipStream_t* getNewStream(void){
    hipStream_t* stream = (hipStream_t*)malloc(sizeof(hipStream_t));
    CudaSafeAPICall(hipStreamCreate(stream));
    return stream;
}

void cleanUpStream(hipStream_t* stream){
    CudaSafeAPICall(hipStreamDestroy(*stream));
}


int getNumberOfCUDADevices(void){
    int dev_count;
    CudaSafeAPICall(hipGetDeviceCount(&dev_count));
    return dev_count;
}

void getDeviceProperties(hipDeviceProp_t* props, int deviceID){
    CudaSafeAPICall(hipGetDeviceProperties(props, deviceID));
}

