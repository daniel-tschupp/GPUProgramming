#include "kernel_utils.h"

float* pinHostAndMallocDeviceMemory(float* ptr, size_t size){
    float *devPtr = nullptr;

    hipHostRegister(ptr,static_cast<uint>(size), hipHostRegisterPortable|hipHostRegisterMapped);
    hipMalloc((void**)&devPtr, size);

    return devPtr;
}
hipError_t cpyAsyncToDevice(float* devPtr, float* hostPtr, size_t size, hipStream_t* stream){
    return hipMemcpyAsync(devPtr, hostPtr, size, H2D, *stream);
}

hipError_t cpyAsyncToHost(float* devPtr, float* hostPtr, size_t size, hipStream_t* stream){
    return hipMemcpyAsync(hostPtr, devPtr, size, D2H, *stream);
}

void unpinHostAndFreeDeviceMemory(float* devPtr, float* hostPtr){
    hipHostUnregister(hostPtr);
    hipFree(devPtr);
}

void waitForStream(hipStream_t* stream){
    hipStreamSynchronize(*stream);
}

hipStream_t* getNewStream(void){
    hipStream_t* stream = (hipStream_t*)malloc(sizeof(hipStream_t));
    hipStreamCreate(stream);
    return stream;
}

void cleanUpStream(hipStream_t* stream){
    hipStreamDestroy(*stream);
}


int getNumberOfCUDADevices(void){
    int dev_count;
    hipGetDeviceCount(&dev_count);
    return dev_count;
}

void getDeviceProperties(hipDeviceProp_t* props, int deviceID){
    hipGetDeviceProperties(props, deviceID);
}
