#include "kernel_utils.h"

void pinHostMemory(float* hostPtr, size_t size){
    hipHostRegister(hostPtr,static_cast<uint>(size), hipHostRegisterPortable|hipHostRegisterMapped);
}
float* mallocDeviceMemory(size_t size){
    float *devPtr = nullptr;
    hipMalloc((void**)&devPtr, size);
    return devPtr;
}

float* pinHostAndMallocDeviceMemory(float* ptr, size_t size){
    float *devPtr = nullptr;
    hipHostRegister(ptr,static_cast<uint>(size), hipHostRegisterPortable|hipHostRegisterMapped);
    hipMalloc((void**)&devPtr, size);
    return devPtr;
}
hipError_t cpyAsyncToDevice(float* devPtr, float* hostPtr, size_t size, hipStream_t* stream){
    return hipMemcpyAsync(devPtr, hostPtr, size, H2D, *stream);
}

hipError_t cpyAsyncToHost(float* devPtr, float* hostPtr, size_t size, hipStream_t* stream){
    return hipMemcpyAsync(hostPtr, devPtr, size, D2H, *stream);
}

void unpinHostMemory(float* hostPtr){
    hipHostUnregister(hostPtr);
}
void freeDeviceMemory(float* devPtr){
    hipFree(devPtr);
}
void unpinHostAndFreeDeviceMemory(float* devPtr, float* hostPtr){
    hipHostUnregister(hostPtr);
    hipFree(devPtr);
}

void waitForStream(hipStream_t* stream){
    hipStreamSynchronize(*stream);
}

hipStream_t* getNewStream(void){
    hipStream_t* stream = (hipStream_t*)malloc(sizeof(hipStream_t));
    hipStreamCreate(stream);
    return stream;
}

void cleanUpStream(hipStream_t* stream){
    hipStreamDestroy(*stream);
}


int getNumberOfCUDADevices(void){
    int dev_count;
    hipGetDeviceCount(&dev_count);
    return dev_count;
}

void getDeviceProperties(hipDeviceProp_t* props, int deviceID){
    hipGetDeviceProperties(props, deviceID);
}
